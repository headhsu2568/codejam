#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cutil_inline.h>
#include <time.h>
#include <assert.h>
#include <sm_11_atomic_functions.h>
#include <shrUtils.h>


struct position_t
{
    int lock;
    int diff;
    int width;
    int height;
};

int frame_width = 1024;
int frame_height = 1024;

int body_width = 32;
int body_height = 32;

int* h_frame = NULL;
int* h_body = NULL;
int* h_diff = NULL;
position_t* h_position = NULL;

int* d_frame = NULL;
int* d_body = NULL;
int* d_diff = NULL;
position_t* d_position = NULL;

int LOOP_NUM = 1;

unsigned int seed = 0x1234567;

//timespec start_time;
//timespec end_time;

__global__ void body_track(int* frame, int frame_width, int frame_height,
        int* body, int body_width, int body_height, int* diff, position_t* pos)
{

};

unsigned int myrand(unsigned int *seed, unsigned int input)
{
    *seed ^= (*seed << 13) ^ (*seed >> 15) + input;
    *seed += (*seed << 17) ^ (*seed >> 14) ^ input;
    return *seed;
};

void sig_check()
{
    unsigned int sig = 0x1234567;
    for(int i = 0; i < frame_height; i++)
        for(int j = 0; j < frame_width; j++)
            myrand(&sig, h_diff[i*frame_width+j]);

    //myrand(&sig, h_position->height);
    //myrand(&sig, h_position->width);

    printf("Computed check sum signature:0x%08x\n", sig);
    if(sig == 0x17dd3971)
        printf("Result check by signature successful!!\n");
    else
        printf("Result check by signature failed!!\n");
}

void show_array(int* array, int width, int height)
{
    for(int i = 0; i < height; i++)
    {
        for(int j = 0; j < width; j++)
            printf("%03d, ", array[i*width+j]);
        printf("\n");
    }
    printf("\n");
}

int main (int argc, char *argv[])
{
    // get the dimension of array
    assert(argc == 2);
    LOOP_NUM = atoi(argv[1]);

    printf("LOOP_NUM:%d\n", LOOP_NUM);

    // Allocate input vectors h_A and h_B in host memory
    h_frame = (int*)malloc(frame_width*frame_height*sizeof(int));
    h_body = (int*)malloc(body_width*body_height*sizeof(int));
    h_diff = (int*)malloc(frame_width*frame_height*sizeof(int));
    h_position = (position_t*)malloc(sizeof(position_t));
    assert(h_frame);
    assert(h_body);
    assert(h_diff);
    assert(h_position);

    // initial frame, body, diff
    for(int i = 0; i < frame_height; i++)
        for(int j = 0; j < frame_width; j++)
        {
            h_frame[i*frame_width+j] = myrand(&seed, i*j) & 0xff;
            h_diff[i*frame_width+j] = 0;
        }

    for(int i = 0; i < body_height; i++)
        for(int j = 0; j < body_width; j++)
        {
            h_body[i*body_width+j] = myrand(&seed, i*j) & 0xff;
        }

    h_position->lock = 0;
    h_position->diff = 0x7fffffff;
    h_position->width = -1;
    h_position->height = -1;

    //clock_gettime(CLOCK_REALTIME, &start_time);

    // Allocate vectors in device memory
    cutilSafeCall( hipMalloc((void**)&d_frame, frame_width*frame_height*sizeof(int)) );
    cutilSafeCall( hipMalloc((void**)&d_body, body_width*body_height*sizeof(int)) );
    cutilSafeCall( hipMalloc((void**)&d_diff, frame_width*frame_height*sizeof(int)) );
    cutilSafeCall( hipMalloc((void**)&d_position, sizeof(*h_position)) );

    // Copy vectors from host memory to device memory
    cutilSafeCall( hipMemcpy(d_frame, h_frame, frame_width*frame_height*sizeof(int), hipMemcpyHostToDevice) );
    cutilSafeCall( hipMemcpy(d_body, h_body, body_width*body_height*sizeof(int), hipMemcpyHostToDevice) );
    cutilSafeCall( hipMemcpy(d_position, h_position, sizeof(*h_position), hipMemcpyHostToDevice) );

    // Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (frame_height*frame_width + threadsPerBlock - 1) / threadsPerBlock;
    body_track<<<blocksPerGrid, threadsPerBlock>>>(d_frame, frame_width, frame_height, d_body, body_width, body_height, d_diff, d_position);
    cutilCheckMsg("kernel launch failure");

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    cutilSafeCall( hipMemcpy(h_diff, d_diff, frame_width*frame_height*sizeof(int), hipMemcpyDeviceToHost) );
    cutilSafeCall( hipMemcpy(h_position, d_position, sizeof(*h_position), hipMemcpyDeviceToHost) );

    //clock_gettime(CLOCK_REALTIME, &end_time);

    printf("position(%d,%d):%d\n", h_position->width, h_position->height, h_position->diff);
    //printf("sizeof(start_time.tv_sec):%d, sizeof(start_time.tv_nsec):%d\n", sizeof(start_time.tv_sec), sizeof(start_time.tv_nsec));
    //printf("s_time.tv_sec:%d, s_time.tv_nsec:%d\n", start_time.tv_sec, start_time.tv_nsec);
    //printf("e_time.tv_sec:%d, e_time.tv_nsec:%d\n", end_time.tv_sec, end_time.tv_nsec);
    //double execution_time = (double)end_time.tv_sec + (double)end_time.tv_nsec/1000000000.0
    //  - (double)start_time.tv_sec - (double)start_time.tv_nsec/1000000000.0;
    //printf("diff_time:%.4f(s)\n", execution_time);

    //show_array(h_frame, frame_width, frame_height);
    //show_array(h_body, body_width, body_height);
    //show_array(h_diff, frame_width, frame_height);

    sig_check();

    //cutilSafeCall( hipDeviceReset() );

    return 0;
}
