#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <sys/time.h>
#include <sys/times.h>
#include <sys/resource.h>
#include <errno.h>
#include <unistd.h>
#include <cutil_inline.h>

// Variables
int* h_A = NULL;
int* h_B = NULL;
int* h_C = NULL;
int* d_A = NULL;
int* d_B = NULL;
int* d_C = NULL;

int N = 10;  
unsigned int seed = 0x1234567;

timespec start_time;                                 
timespec end_time;                                 

void array_mul(int* A, int* B, int* C, int N) 
{        
    for(int i = 0; i < N; i++) 
    {
        for(int j = 0; j < N; j++) 
        {          
            for(int k = 0; k < N; k++) 
            {
                C[i*N+j] = C[i*N+j] + A[i*N+k] * B[k*N+j]; 
            }  
        }      
    }
};

__global__ void MatrixMulKernel(int* d_A, int* d_B, int* d_C, int width){
    int Cvalue=0;
    for(int k=0; k<width; ++k){
        int Aelement = d_A[blockIdx.x*width+k];
        int Belement = d_B[k*width+threadIdx.y];
        Cvalue += Aelement * Belement;
    }
    d_C[blockIdx.x*width+threadIdx.y] = Cvalue;
}

unsigned int myrand(unsigned int *seed, unsigned int input)
{  
    *seed = (*seed << 13) ^ (*seed >> 15) + input + 0xa174de3;
    return *seed;
};

void sig_check()
{    
    unsigned int sig = 0x1234567;
    for(int i = 0; i < N; i++)
    {    
        myrand(&sig, h_C[i]);    
    }           

    printf("Computed check sum signature:0x%08x\n", sig);
    if(sig == 0x9f3afc72)
        printf("Result check by signature successful!!\n");
    else
        printf("Result check by signature failed!!\n");
}

void show_array(int* array)
{
    for(int i = 0; i < N; i++)
    {
        for(int j = 0; j < N; j++)
            printf("%13d, ", array[i*N+j]);
        printf("\n");  
    }
    printf("\n");
}

int main (int argc, char *argv[])
{ 
    // get the dimension of array    
    assert(argc == 2);  
    N = atoi(argv[1]);  
    int size = N*N*sizeof(int);

    printf("N:%d, size:%d\n", N, size);

    // Allocate input vectors h_A and h_B in host memory
    h_A = (int*)malloc(size);  
    h_B = (int*)malloc(size);  
    h_C = (int*)malloc(size);
    assert(h_A);  
    assert(h_B);  
    assert(h_C);  

    // initial array A & B
    for(int i = 0; i < N; i++)
        for(int j = 0; j < N; j++)
        {   
            h_A[i*N+j] = myrand(&seed, i*j) & 0xff;
            h_B[i*N+j] = myrand(&seed, i*j) & 0xff;
            h_C[i*N+j] = 0; 
        }


    // Allocate and copy
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    clock_gettime(CLOCK_REALTIME, &start_time);

    // Invoke kernel
    dim3 dimGrid(1, 1);
    dim3 dimBlock(N, N);

    // Launch the device computation
    MatrixMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    //array_mul(h_A, h_B, h_C, N);    

    clock_gettime(CLOCK_REALTIME, &end_time);    

    printf("sizeof(start_time.tv_sec):%d, sizeof(start_time.tv_nsec):%d\n", sizeof(start_time.tv_sec), sizeof(start_time.tv_nsec));
    printf("s_time.tv_sec:%d, s_time.tv_nsec:%d\n", start_time.tv_sec, start_time.tv_nsec);
    printf("e_time.tv_sec:%d, e_time.tv_nsec:%d\n", end_time.tv_sec, end_time.tv_nsec);
    double execution_time = (double)end_time.tv_sec + (double)end_time.tv_nsec/1000000000.0 
        - (double)start_time.tv_sec - (double)start_time.tv_nsec/1000000000.0;
    printf("diff_time:%.4f(s)\n", execution_time);

    //show_array(h_A);
    //show_array(h_B);
    //show_array(h_C);

    sig_check();         

    return 0;
}
